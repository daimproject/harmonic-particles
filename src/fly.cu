// Fly ions through multipole fields
// vim:set ts=2 sts=2 sw=2 expandtab:

#include <iostream>

#include <thrust/device_vector.h>
//#include <thrust/reduce.h>
//#include <thrust/functional.h>
#include <boost/numeric/odeint.hpp>

// When compiled with nvcc
#ifdef __HIPCC__
#include <hip/hip_runtime_api.h>
void CleanUp(){ hipProfilerStop();}
#else
void CleanUp(){ }
#endif

using namespace boost::numeric::odeint;

// State types
typedef double value_type;
typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< size_t > index_vector_type;

// Constants -- units are SI
const size_t N = 10;
const value_type dx = 1;
const value_type dt = 0.01;

state_type new_state_vector( size_t N, size_t i )
{
  state_type state_vector( N );
  thrust::fill(state_vector.begin(), state_vector.end(), i);
  return state_vector;
}

int main( int arc , char* argv[] )
{
  std::cout << "Hello \n";

  state_type x = new_state_vector( 10, 8 );
  
  for( size_t i=0; i<N; ++i ){
    std::cout << x[i]; 
  }

  
  CleanUp();
  return 0;
}
